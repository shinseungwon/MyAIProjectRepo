
#include <hip/hip_runtime.h>
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//
//#include <stdio.h>
//#include <iostream>
//#include <vector>
//#include <fstream>
//#include <string>
//#include <algorithm>
//#include <chrono>
//#include <random>
//
//using namespace std;
//
//#define IMAGE_PATH "mnist\\train-images.idx3-ubyte"
//#define LABEL_PATH "mnist\\train-labels.idx1-ubyte"
//#define BIAS_WEIGHT_PATH "mnist\\biasweight.txt"
//
//#define IMAGE_SIZE 28
//#define IMAGE_HEADER_SIZE 16
//#define LABEL_HEADER_SIZE 8
//
//#define TOTAL_SIZE 1200 //60000 max
//#define MINI_BATCH_SIZE 100 // <= TOTAL_IMG_SIZE
//
//#define LEARN_COUNT 20
//#define LEARN_RATE 1
//#define DIFF_H 1e-4;
//#define DIFF_H_2 2 * 1e-4;
//
//struct Image {
//	int num;
//	unsigned char image[28][28];
//};
//
//float sigmoid_cu(float x);
//vector<float>* softmax_cu(vector<float>* a);
//float* matrix_multiplication_cu(float* a, int m, int n, float* b, int o, int p, int blocks, int threads);
//float** matrix_transpose_cu(float** a, int m, int n);
//void set_sigmoid_backward_cu(float** dy, float m, float n, float** y, float o, float p);
//
//vector<Image>* read_image_label_cu(const char* image_path, const char* label_path);
//vector<vector<float>*>* read_bias_weight_cu(const char* bias_weight_path);
//void write_bias_weight_cu(const char* path, float** bs, int* m, float*** ws, int** n);
//
//void print_image_cu(Image* img);
//void print_matrix_cu(float** a, int m, int n);
//float* convert_image_cu(Image* img);
//
//float* get_normal_distribution_array_cu(int n);
//vector<int>* mini_batch_idx_sort_cu(int n, int count);
//float cross_entropy_error_cu(vector<float>* y, vector<float>* t);
//
//vector<float>* predict_cu(Image* img, vector<vector<float>*>* bs, vector<vector<vector<float>*>*>* ws);
//float*** predictlevels_cu(Image* img, float** bs, float*** ws);
//float*** get_weight_gradient(Image* img, float** bs, float*** ws);
//
//void learn_cu(Image* img, float** bs, float*** ws);
//void backprop_cu(Image* img, float** bs, float*** ws);
//void printpredict_cu(vector<Image>* images, vector<int>* mini_batch, vector<vector<float>*>* bs, vector<vector<vector<float>*>*>* ws);
//
//__global__ void kernel_mm(float* a, int m, int n, float* b, int o, int p, float* res);
//__global__ void kernel_bs(float* a, int m, int n, float* bs);
//
//vector<vector<float>*>* bv_cu;
//vector<vector<vector<float>*>*>* wv_cu;
//vector<vector<float>*>* bg_cu;
//vector<vector<vector<float>*>*>* wg_cu;
//
//int main()
//{
//	//initialize
//	srand(static_cast<unsigned int>(time(NULL)));
//	vector<Image>* images = read_image_label_cu(IMAGE_PATH, LABEL_PATH);
//	vector<vector<float>*>* vs = read_bias_weight_cu("mnist\\bwvalue.txt");
//
//	vector<vector<float>*>* bs = new vector<vector<float>*>();
//	int i, j, k, l, cnt = 0;
//	for (i = 0; i < 3; i++) {
//		bs->push_back(vs->at(cnt++));		
//	}
//
//	vector<vector<vector<float>*>*>* ws = new vector<vector<vector<float>*>*>();
//	vector<vector<float>*>* wt;
//
//	wt = new vector<vector<float>*>();
//	for (i = 0; i < 784; i++) {
//		wt->push_back(vs->at(cnt++));
//		//wt->push_back(get_normal_distribution_array(50));
//		//wt->push_back(new vector<float>(50, 0.5));
//	}
//	ws->push_back(wt);
//
//	wt = new vector<vector<float>*>();
//	for (i = 0; i < 50; i++) {
//		wt->push_back(vs->at(cnt++));
//		//wt->push_back(get_normal_distribution_array(100));
//		//wt->push_back(new vector<float>(100, 0.5));
//	}
//	ws->push_back(wt);
//
//	wt = new vector<vector<float>*>();
//	for (i = 0; i < 100; i++) {
//		wt->push_back(vs->at(cnt++));
//		//wt->push_back(get_normal_distribution_array(10));
//		//wt->push_back(new vector<float>(10, 0.5));
//	}
//	ws->push_back(wt);
//	//~initialize
//
//	//cnn
//	int n = images->size(), correct = 0, wrong = 0, seq = 0, max_index;
//	float error, max_value, mh, ph;
//
//	vector<int>* mini_batch = mini_batch_idx_sort_cu(TOTAL_SIZE, MINI_BATCH_SIZE);
//
//	//learn
//	printpredict_cu(images, mini_batch, bs, ws);
//	//for (i = 0; i < MINI_BATCH_SIZE; i++) {
//	//	cout << i << " - learn " << images->at(mini_batch->at(i)).num << ' ';
//	//	//learn(&images->at(mini_batch->at(i)), bs, ws);
//	//	backprop_cu(&images->at(mini_batch->at(i)), bs, ws);
//	//	cout << endl;
//	//}
//	printpredict_cu(images, mini_batch, bs, ws);
//	//~learn
//
//	//~cnn
//
//	//deallocate
//	delete mini_batch;
//	for (i = 0; i < ws->size(); i++) {
//		delete ws->at(i);
//	}
//	delete ws;
//
//	for (i = 0; i < vs->size(); i++) {
//		delete vs->at(i);
//	}
//	delete vs;
//	delete images;
//	//~deallocate
//
//	return 0;
//}
//
//vector<Image>* read_image_label_cu(const char* image_path, const char* label_path)
//{
//	int header = 0, row = 0, col = 0, n, m, i = 0, j = 0, k = 0, l = 0, count = 0;
//
//	//read image
//	ifstream input_image(image_path, ios::binary);
//	//vector<char> bytes_i(istreambuf_iterator<char>(input_image), (istreambuf_iterator<char>()));
//	vector<char> bytes_i;
//	char headerbuffer[IMAGE_HEADER_SIZE];
//	input_image.read(headerbuffer, IMAGE_HEADER_SIZE);
//	for (i = 0; i < IMAGE_HEADER_SIZE; i++) {
//		bytes_i.push_back(headerbuffer[i]);
//	}
//
//	for (i = 0; i < TOTAL_SIZE; i++) {
//		char imagebuffer[784];
//		input_image.read(imagebuffer, 784);
//		for (j = 0; j < 784; j++) {
//			bytes_i.push_back(imagebuffer[j]);
//		}
//	}
//
//	//read label
//	ifstream input_label(label_path, ios::binary);
//	//vector<char> bytes_l(istreambuf_iterator<char>(input_label), (istreambuf_iterator<char>()));
//	vector<char> bytes_l;
//	char labelbuffer[LABEL_HEADER_SIZE + TOTAL_SIZE];
//	input_label.read(labelbuffer, LABEL_HEADER_SIZE + TOTAL_SIZE);
//	for (int i = 0; i < LABEL_HEADER_SIZE + TOTAL_SIZE; i++) {
//		bytes_l.push_back(labelbuffer[i]);
//	}
//
//	n = bytes_i.size();
//	m = bytes_l.size();
//	vector<Image>* res = new vector<Image>();
//
//	i = 0;
//	l = 0;
//	cout << "image header : ";
//	for (j = 0; j < IMAGE_HEADER_SIZE; j++) {
//		cout << (int)(unsigned char)bytes_i[i++] << ' ';
//	}
//	cout << endl;
//
//	cout << "label header : ";
//	for (j = 0; j < LABEL_HEADER_SIZE; j++) {
//		cout << (int)(unsigned char)bytes_l[l++] << ' ';
//	}
//	cout << endl;
//
//	while (i < n && l < m && count < TOTAL_SIZE) {
//
//		Image img;
//		img.num = (int)(unsigned char)bytes_l[l++];
//
//		for (j = 0; j < IMAGE_SIZE; j++) {
//			for (k = 0; k < IMAGE_SIZE; k++) {
//				img.image[j][k] = (unsigned char)bytes_i[i++];
//			}
//		}
//		res->push_back(img);
//		//print_image(&img);
//		count++;
//	}
//
//	cout << res->size() << " images" << endl;
//
//	input_image.close();
//	input_label.close();
//
//	return res;
//}
//
//vector<vector<float>*>* read_bias_weight_cu(const char* bias_weigh_tpath) {
//	ifstream bias_wieght(bias_weigh_tpath);
//	vector<char> chars(istreambuf_iterator<char>(bias_wieght), (istreambuf_iterator<char>()));
//	int n = chars.size(), i, k;
//	vector<vector<float>*>* vs = new vector<vector<float>*>();
//	vector<float>* v = new vector<float>();
//	string s = "";
//	for (i = 0; i < n; i++) {
//		if (chars[i] == '\n') {
//			vs->push_back(v);
//			v = new vector<float>();
//		}
//		else if (chars[i] == '/') {
//			v->push_back(stof(s));
//			s.clear();
//		}
//		else {
//			s += chars[i];
//		}
//	}
//
//	return vs;
//}
//
//vector<int>* mini_batch_idx_sort_cu(int n, int count) {
//	vector<int>* res = new vector<int>();
//	int i, x;
//
//	if (count <= n && count >= 0) {
//		int* arr = new int[n];
//		for (i = 0; i < n; i++) {
//			arr[i] = i;
//		}
//		for (i = n; i > n - count; i--) {
//			x = rand() % i;
//			res->push_back(arr[x]);
//			arr[x] = arr[i - 1];
//		}
//		delete[] arr;
//	}
//	sort(res->begin(), res->end());
//	return res;
//}
//
//
//void printpredict_cu(vector<Image>* images, vector<int>* mini_batch, float** bs, int* bs_arr_sz, int bs_sz
//	, vector<vector<vector<float>*>*>* ws) {
//	int i, j, seq, max_index, correct = 0, wrong = 0;
//	float error, max_value;
//	for (i = 0; i < MINI_BATCH_SIZE; i++) {
//		seq = mini_batch->at(i);
//		cout << "task " << i + 1 << " -> value : " << images->at(seq).num;
//		vector<float>* output = predict_cu(&images->at(seq), bs, ws);
//		vector<float>* answer = new vector<float>(10, 0);
//		answer->at(images->at(seq).num) = 1;
//		error = cross_entropy_error_cu(output, answer);
//
//		max_value = 0;
//		max_index = -1;
//		for (j = 0; j < output->size(); j++) {
//			if (output->at(j) > max_value) {
//				max_value = output->at(j);
//				max_index = j;
//			}
//		}
//
//		if (max_index == images->at(seq).num) {
//			correct++;
//		}
//		else {
//			wrong++;
//		}
//		cout << " result : " << max_index << " error : " << error
//			<< " accuracy : " << correct / (float)(correct + wrong) << endl;
//
//		delete output;
//		delete answer;
//	}	
//}
//
//float* convert_image_cu(Image* image) {
//	float* res = new float[IMAGE_SIZE * IMAGE_SIZE];
//	int i, j;
//	for (i = 0; i < IMAGE_SIZE; i++) {
//		for (j = 0; j < IMAGE_SIZE; j++) {
//			res[i * IMAGE_SIZE + j] = image->image[i][j];
//		}
//	}
//	return res;
//}
//
//vector<float>* predict_cu(Image* img, vector<vector<float>*>* bs, vector<vector<vector<float>*>*>* ws) {
//	float* input = convert_image_cu(img);	
//	int i;
//
//	float* ws0 = ws->at(0)
//	
//	float* level_1 = matrix_multiplication_cu(input, 28, 28,  );
//	set_bias_sigmoid_cu(level_1, bs->at(0));
//
//	float* level_2 = matrix_multiplication_cu(level_1, ws->at(1));
//	set_bias_sigmoid_cu(level_2, bs->at(1));
//
//	float* level_3 = matrix_multiplication_cu(level_2, ws->at(2));
//	set_bias_sigmoid_cu(level_3, bs->at(2));
//
//	vector<float>* output = softmax_cu(level_3->at(0));
//
//	delete level_1;
//	delete level_2;
//	delete level_3;	
//	delete input;
//
//	return output;
//}
//
//vector<float>* softmax_cu(vector<float>* a) {
//	int i, n = a->size();
//	float max = 0, expsum = 0;
//	vector<float> ac(n);
//	vector<float>* res = new vector<float>();
//	for (i = 0; i < n; i++) {
//		if (a->at(i) > max) {
//			max = a->at(i);
//		}
//	}
//
//	for (i = 0; i < n; i++) {
//		ac[i] = a->at(i) - max;
//		expsum += exp(ac[i]);
//	}
//
//	for (i = 0; i < n; i++) {
//		res->push_back(exp(ac[i]) / expsum);
//	}
//
//	return res;
//}
//
//float* matrix_multiplication_cu(float* a, int m, int n, float* b, int o, int p, float* bs, int q, int blocks, int threads) {
//
//	float* dev_a = 0;
//	float* dev_b = 0;
//	float* dev_c = 0;
//
//	int mal_a = m * n;
//	int mal_b = o * p;
//	int mal_c = m * p;
//
//	cudaMalloc((void**)&dev_c, mal_c * sizeof(float));
//
//	cudaMalloc((void**)&dev_a, mal_a * sizeof(float));
//	cudaMemcpy(dev_a, a, mal_a * sizeof(float), cudaMemcpyHostToDevice);	
//
//	cudaMalloc((void**)&dev_b, mal_b * sizeof(float));
//	cudaMemcpy(dev_b, b, mal_b * sizeof(float), cudaMemcpyHostToDevice);
//
//	kernel_mm << <blocks, threads >> > (a, m, n, b, o, p, c);
//
//	cudaMemcpy(dev_c, dev_c, mal_c * sizeof(int), cudaMemcpyDeviceToHost);
//
//	return dev_c;
//}
//
//__global__ void kernel_mm(float* a, int m, int n, float* b, int o, int p, float* res) {
//	
//}
//
//__global__ void kernel_bs(float* a, int m, int n, float* bs) {
//
//}