#include "hip/hip_runtime.h"
#include "tools.cuh"

using namespace std;

float* mal_cpy_dev(float* data, int size) {
	float* res = mal_dev(size);
	cpy_dev(res, data, size);
	return res;
}

float* mal_cpy_host(float* data, int size) {
	float* res = new float[size];
	cpy_host(res, data, size);
	return res;
}

float* mal_dev(int size) {
	float* res;

	hipError_t error;
	error = hipMalloc((void**)&res, size * sizeof(float));
	if (error != hipSuccess) {
		throw new exception("Cuda Malloc Failed");
	}

	error = hipMemset(res, 0, size * sizeof(float));
	if (error != hipSuccess) {
		throw new exception("Cuda Memset Failed");
	}

	return res;
}

void free_dev(float* data) {
	hipError_t error;
	error = hipFree(data);
	if (error != hipSuccess) {
		throw new exception("Cuda Free Failed");
	}
}

void cpy_host(float* dst, float* src, int size) {
	hipError_t error;
	error = hipMemcpy(dst, src, size * sizeof(float), hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		throw new exception("Cuda Memcpy Failed");
	}
}

void cpy_dev(float* dst, float* src, int size) {
	hipError_t error;
	error = hipMemcpy(dst, src, size * sizeof(float), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		throw new exception("Cuda Memcpy Failed");
	}
}

void cpy_dev_to_dev(float* dst, float* src, int size) {
	hipError_t error;
	error = hipMemcpy(dst, src, size * sizeof(float), hipMemcpyDeviceToDevice);
	if (error != hipSuccess) {
		throw new exception("Cuda Memcpy Failed");
	}
}

void generate_weight(float* weight, int size, float mean, float dist) {
	normal_distribution<float> distribution(mean, dist);
	default_random_engine generator;
	generator.seed(rand());
	float* weight_gen = new float[size];
	for (int i = 0; i < size; i++) {
		weight_gen[i] = distribution(generator);
	}
	cpy_dev(weight, weight_gen, size);
}

//<<<m, p>>> matrix, {n == o} (m x n) x (o x p) = (m x p)
void matrix_multiplication(float* a, int m, int n, float* b, int o, int p, float* res) {

	if (res == nullptr) {
		res = mal_dev(m * p);
	}

	k_matrix_multiplication << <m, p >> > (a, m, n, b, o, p, res);
	hipError_t error = hipDeviceSynchronize();
	if (error != hipSuccess) {
		throw new exception("Function 'matrix_multiplication' Failed");
	}
}

void matrix_convolution_multiplication(float* a, int m, float* b, int n, float* c) {
	int dst_width = m - n + 1;
	k_matrix_convolution_multiplication<<<dst_width, dst_width>>>(a, m, b, n, c);
	hipError_t error = hipDeviceSynchronize();
	if (error != hipSuccess) {
		throw new exception("Function 'matrix_convolution_multiplication' Failed");
	}
}

void matrix_transpose(float* a, int m, int n, float* b) {
	k_matrix_transpose<<<m, n>>>(a, m, n, b);
	hipError_t error = hipDeviceSynchronize();
	if (error != hipSuccess) {
		throw new exception("Function 'matrix_transpose' Failed");
	}
}

void matrix_reverse(float* a, float* b, int m, int n) {
	k_matrix_reverse<<<n, m>>>(a, b, m);
	hipError_t error = hipDeviceSynchronize();
	if (error != hipSuccess) {
		throw new exception("Function 'matrix_reverse' Failed");
	}
}

void make_padding_matrix(float* a, int n, float* b, int p) {
	k_make_padding_matrix<<<n, n>>>(a, n, b, p);
	hipError_t error = hipDeviceSynchronize();
	if (error != hipSuccess) {
		throw new exception("Function 'make_padding_matrix' Failed");
	}
}

void update_delta(float* weight, float* changes, int w, int h, float lr) {
	k_update_delta<<<w, h>>>(weight, changes, w, lr);
	hipError_t error = hipDeviceSynchronize();
	if (error != hipSuccess) {
		cout << "Function 'update_delta' Failed" << endl;
		throw exception(hipGetErrorString(error));
	}
}

void add_bias(int n, float* dst, float* bias) {
	k_add_bias<<<1, n>>>(n, dst, bias);
	hipError_t error = hipDeviceSynchronize();
	if (error != hipSuccess) {
		cout << "Function 'add_bias' Failed" << endl;
		throw exception(hipGetErrorString(error));
	}
}

void add_bias_array(int n, float* dst, float* bias) {
	k_add_bias_array<<<1, n>>>(n, dst, bias);
	hipError_t error = hipDeviceSynchronize();
	if (error != hipSuccess) {
		cout << "Function 'add_bias_array' Failed" << endl;
		throw exception(hipGetErrorString(error));
	}
}

void relu(int n, float* a) {
	k_relu<<<1, n>>>(n, a);
	hipError_t error = hipDeviceSynchronize();
	if (error != hipSuccess) {
		cout << "Function 'relu' Failed" << endl;
		throw exception(hipGetErrorString(error));
	}
}

void relu_backward(float* dy, float* y, int m, int n) {
	k_relu_backward<<<m, n>>>(dy, y, m, n);
	hipError_t error = hipDeviceSynchronize();
	if (error != hipSuccess) {
		cout << "Function 'relu_backward' Failed" << endl;
		throw exception(hipGetErrorString(error));
	}
}

void pooling(float* a, int m, int n, float* b) {
	int pooling_size = m / n;
	k_pooling<<<pooling_size, pooling_size>>>(a, m, n, b);
	hipError_t error = hipDeviceSynchronize();
	if (error != hipSuccess) {
		cout << "Function 'pooling' Failed" << endl;
		throw exception(hipGetErrorString(error));
	}
}

void pooling_backward(float* d, float* a, int m, float* b, int n, float* c) {
	k_pooling_backward<<<n, n>>>(d, a, m, b, n, c);
	hipError_t error = hipDeviceSynchronize();
	if (error != hipSuccess) {
		cout << "Function 'pooling_backward' Failed" << endl;
		throw exception(hipGetErrorString(error));
	}
}

void get_dist_worker(int size, float avg, float* data, float* data_sub_avg, float* data_sub_avg_sq) {
	//��� ���ְ� ���� ���ְ�
	k_get_dist_worker<<<1, size>>>(size, avg, data, data_sub_avg, data_sub_avg_sq);
	hipError_t error = hipDeviceSynchronize();
	if (error != hipSuccess) {
		cout << "Function 'get_dist_worker' Failed" << endl;
		throw exception(hipGetErrorString(error));
	}
}

void batch_norm_worker(int size
	, float* data
	, float* data_sub_avg
	, float* data_sub_avg_sq
	, float* data_caret
	, float* data_caret_mul_g
	, float* data_caret_mul_g_add_b
	, float avg, float dist, float g, float b) {
	float dist_sqrt = sqrt(dist + 10e-7);
	k_batch_norm_worker<<<1, size>>>(size, data_sub_avg, data_caret, data_caret_mul_g, data_caret_mul_g_add_b, dist_sqrt, g, b);
}

void batch_norm(float* a, int size, float avg, float disp, float g, float b) {
	k_batch_norm<<<1, size>>>(a, avg, disp, g, b);
	hipError_t error = hipDeviceSynchronize();
	if (error != hipSuccess) {
		cout << "Function 'batch_norm' Failed" << endl;
		throw exception(hipGetErrorString(error));
	}
}

void dropout(float* data, float* mask, int size) {
	k_dropout<<<1, size>>>(data, mask);
	hipError_t error = hipDeviceSynchronize();
	if (error != hipSuccess) {
		cout << "Function 'dropout' Failed" << endl;
		throw exception(hipGetErrorString(error));
	}
}

void softmax(float* a, float* b, int n) {
	int i;
	float max = 0, expsum = 0;
	for (i = 0; i < n; i++) {
		if (a[i] > max) {
			max = a[i];
		}
		b[i] = a[i];
	}
	for (i = 0; i < n; i++) {
		b[i] -= max;
		expsum += exp(b[i]);
	}

	for (i = 0; i < n; i++) {
		b[i] = exp(b[i]) / expsum;
	}
}

float get_cross_entrophy_error(float* answer, float* output, int size) {
	float res = 0;
	int i;

	for (i = 0; i < size; i++) {
		res += answer[i] * log(output[i] + 1e-7);
	}

	return -res;
}

int* mini_batch(int n, int count) {
	int i, x;
	vector<int>* mbatch = new vector<int>();
	int* arr = new int[n];
	for (i = 0; i < n; i++) {
		arr[i] = i;
	}
	for (i = n; i > n - count; i--) {
		x = rand() % i;
		mbatch->push_back(arr[x]);
		arr[x] = arr[i - 1];
	}
	delete[] arr;
	return &mbatch->at(0);
}

float* mini_batch_mask(int n, int count) {
	int i;
	float* res = new float[n];
	for (i = 0; i < n; i++) {
		res[i] = 1;
	}
	
	int* mbatch = mini_batch(n, count);
	for (int i = 0; i < count; i++) {
		res[mbatch[i]] = 0;		
	}
	return res;
}

void print(char* title, float* data, int size, int width) {
	float* tmp = new float[size];
	cpy_host(tmp, data, size);

	printf("%s (%d items)\n", title, size);
	printf("----------------------------------------------------------------------------------------------------\n");
	for (int i = 0; i < size; i++) {
		if (i % width == 0 && i != 0) {
			printf("(%d)\n", i);
		}
		printf("%.4f ", tmp[i]);
	}
	printf("(%d)\n----------------------------------------------------------------------------------------------------\n", size);
	delete[] tmp;
}

void set_weight_changes(float* weight, float* weight_grad, int h, int w, float lr) {
	k_set_weight_changes<<<h, w>>>(weight, weight_grad, h, w, lr);
	hipError_t error = hipDeviceSynchronize();
	if (error != hipSuccess) {
		cout << "Function 'set_weight_changes' Failed" << endl;
		throw exception(hipGetErrorString(error));
	}
}

void set_bias_changes(float* bias, float* bias_grad, int n, float lr) {
	k_set_bias_changes<<<1, n>>>(bias, bias_grad, n, lr);
	hipError_t error = hipDeviceSynchronize();
	if (error != hipSuccess) {
		cout << "Function 'set_weight_changes' Failed" << endl;
		throw exception(hipGetErrorString(error));
	}
}