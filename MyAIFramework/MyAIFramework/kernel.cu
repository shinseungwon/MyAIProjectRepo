#include "hip/hip_runtime.h"
#include "kernel.cuh"

using namespace std;

//n : col count ( thread count )
__global__ void k_update_delta(float* weight, float* delta, int width, float lr) {
	int idx = blockIdx.x * width + threadIdx.x;
	weight[idx] -= lr * delta[idx];
}

__global__ void k_add_bias(int n, float* a, float* bias) {	
	int idx = blockIdx.x * n + threadIdx.x;
	a[idx] += *bias;
}

__global__ void k_add_bias_array(int n, float* a, float* bias) {
	int idx = blockIdx.x * n + threadIdx.x;
	a[idx] += bias[idx];
}

__global__ void k_relu(int n, float* a) {
	int idx = blockIdx.x * n + threadIdx.x;
	if (a[idx] < 0) a[idx] = 0;
}

__global__ void k_relu_backward(float* dy, float* y, int m, int n) {
	int i = blockIdx.x, j = threadIdx.x, seq = n * i + j;
	dy[seq] *= y[seq] < 0 ? 0 : 1;
}

__global__ void k_pooling(float* a, int m, int n, float* b) {
	int bl = blockIdx.x, th = threadIdx.x;
	int bs = bl * n, ts = th * n, rs = bl * (m / n) + th, tmp;
	int i, j;
	b[rs] = 0;
	for (i = bs; i < bs + n; i++) {
		for (j = ts; j < ts + n; j++) {
			tmp = i * m + j;
			b[rs] = a[tmp] > b[rs] ? a[tmp] : b[rs];
		}
	}
}

//d : delta, a : pooling data, m : pooling size, b : cnn data, n : cnn size, c : pooling backward result
__global__ void k_pooling_backward(float* d, float* a, int m, float* b, int n, float* c) {
	int bl = blockIdx.x, th = threadIdx.x;
	int cnnIdx = bl * n + th;
	int poolIdx = (bl / 2) * m + (th / 2);
	c[cnnIdx] = a[poolIdx] == b[cnnIdx] ? d[poolIdx] : 0;
}

//<<<m, p>>> matrix, {n == o} (m x n) x (o x p) = (m x p)
__global__ void k_matrix_multiplication(float* a, int m, int n, float* b, int o, int p, float* res) {
	int bi = blockIdx.x, ti = threadIdx.x, sb = bi * n, st = ti, c = bi * p + ti, i;
	res[c] = 0;
	for (i = 0; i < n; i++) {
		res[c] += a[sb + i] * b[st];
		st += p;
	}
}

//<<<m - n + 1, m - n + 1>>>
__global__ void k_matrix_convolution_multiplication(float* a, int m, float* b, int n, float* c) {
	int bl = blockIdx.x, th = threadIdx.x;
	int o = m - n + 1, p = bl * o + th;
	int i, j;
	c[p] = 0;
	for (i = bl; i < bl + n; i++) {
		for (j = th; j < th + n; j++) {
			c[p] += a[i * m + j] * b[(i - bl) * n + (j - th)];
		}
	}
}

__global__ void k_matrix_transpose(float* a, int m, int n, float* b) {
	int j = blockIdx.x;
	int i = threadIdx.x;
	b[i * m + j] = a[j * n + i];
}

__global__ void k_matrix_reverse(float* a, float* b, int n) {
	int bl = blockIdx.x, th = threadIdx.x;
	int idx = bl * n + th;
	int ridx = (n - bl - 1) * n + (n - th - 1);
	b[idx] = a[ridx];
}

__global__ void k_make_padding_matrix(float* a, int n, float* b, int p) {
	int bl = blockIdx.x, th = threadIdx.x;
	int idx = bl * n + th;
	int pad_width = n + 2 * p;
	int pad_bl = p - 1 + bl, pad_th = p - 1 + th;
	int pad_idx = pad_bl * pad_width + pad_th;
	b[pad_idx] = a[idx];
}

__global__ void k_set_weight_changes(float* ws, float* wds, int h, int w, float lr) {
	int bx = blockIdx.x;
	int tx = threadIdx.x;
	int seq = bx * w + tx;
	ws[seq] -= wds[seq] * lr;
}

__global__ void k_set_bias_changes(float* bs, float* bds, int n, float lr) {
	int bx = blockIdx.x;
	int tx = threadIdx.x;
	bs[tx] -= bds[tx] * lr;
}

__global__ void k_get_dist_worker(int size, float avg, float* data, float* data_sub_avg, float* data_sub_avg_sq) {
	int bx = blockIdx.x;
	int tx = threadIdx.x;
	data_sub_avg[tx] = data[tx] - avg;
	data_sub_avg_sq[tx] = data_sub_avg[tx] * data_sub_avg[tx];
}

__global__ void k_batch_norm_worker(int size
	, float* data_sub_avg
	, float* data_caret
	, float* data_caret_mul_g
	, float* data_caret_mul_g_add_b
	, float dist_sqrt, float g, float b) {
	int bx = blockIdx.x;
	int tx = threadIdx.x;
	data_caret[tx] = data_sub_avg[tx] / dist_sqrt;
	data_caret_mul_g[tx] = data_caret[tx] * g;
	data_caret_mul_g_add_b[tx] = data_caret_mul_g[tx] + b;
}

__global__ void k_batch_norm(float* a, float avg, float disp, float g, float b) {
	int i = threadIdx.x;
	a[i] = g * ((a[i] - avg) / sqrt(disp * disp + 10e-7)) + b;
}

__global__ void k_dropout(float* data, float* mask) {
	int i = threadIdx.x;
	if (mask[i] == 0) {
		data[i] = 0;
	}
}