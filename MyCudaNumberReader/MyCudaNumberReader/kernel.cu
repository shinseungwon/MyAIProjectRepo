#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdio.h>
#include <fstream>
#include <vector>
#include <string>
#include <chrono>
#include <random>

using namespace std;

//prepare test data
#define IMAGE_PATH "mnist\\train-images.idx3-ubyte"
#define LABEL_PATH "mnist\\train-labels.idx1-ubyte"

#define BIAS_WEIGHT_PATH "mnist\\biasweight.txt" //learned data

#define IMAGE_HEADER_SIZE 16
#define LABEL_HEADER_SIZE 8

#define TOTAL_SIZE 1000 // <= 60000
#define MINI_BATCH_SIZE 100 // <= TOTAL_IMG_SIZE
//~prepare test data

#define LEVEL_SIZE 3

#define LEARN_COUNT 5
#define LEARN_RATE 0.01 // sigmoid : 0.1, relu : 0.000001

//0 : sigmoid, 1 : relu
#define ACT_FUNCTION 1

#define FILTER_SIZE 25 //5 * 5
#define FILTER_WIDTH 5
#define FILTER_COUNT 2
#define POOLING_SIZE 2

vector<float*>* images;
vector<int>* labels;
vector<int>* mbatch;

float* level[LEVEL_SIZE + 1];
int level_size[LEVEL_SIZE + 1]{ 784, 50, 100, 10 };

float** bss;
float** wss;

float* output;
float* output_dev;

//ann
float* bss_dev[LEVEL_SIZE];
float* wss_dev[LEVEL_SIZE];

float* wsst[LEVEL_SIZE];
float* wssd[LEVEL_SIZE];
float* xssd[LEVEL_SIZE + 1];

//cnn
int cnn_level_size[FILTER_COUNT + 1]{ 28, 24, 20 };
float* cnn[FILTER_COUNT];
float* cnn_w[FILTER_COUNT];
float* cnn_b[FILTER_COUNT];
float* cnn_b_w[FILTER_COUNT];

float* filter_gen[FILTER_COUNT];
float* filter[FILTER_COUNT];
float* df[FILTER_COUNT];
float* fr[FILTER_COUNT];

int pad_df_width;
int pad_df_size;
float* pad_df[FILTER_COUNT];

//for accuracy
int cnt = 0;
int correct = 0;

//cross-entrophy
float cee;
//answer value
int anstmp = 0;
//answer array
float* ans;

//batch-norm
float* avg;
float* disp;

hipError_t error;

void read_image_label();
void read_bias_weight();
void make_bias_weight();
void make_cnn_filter();

void mini_batch(int n, int count);
void predict(float* image, int label);
void backprop(float* img, int label);
float cross_entropy_error(float* y, float* t, int n);
void print_matrix(float* a, int n, int r, string tag);

__global__ void softmax(float* a, float* b, int n);
__global__ void matrix_multiplication(float* a, int m, int n, float* b, int o, int p, float* res);
__global__ void bias_sigmoid(float* a, float* bias);
__global__ void bias_relu(float* a, float* bias);
__global__ void sigmoid_backward(float* dy, float* y, int m, int n);
__global__ void relu_backward(float* dy, float* y, int m, int n);
__global__ void relu_cnn(float* a, int n, float* bias);
__global__ void set_weight_changes(float* ws, float* wds, int h, int w);
__global__ void set_bias_changes(float* bs, float* bds, int n);
__global__ void matrix_transpose(float* a, int m, int n, float* b);
__global__ void batch_norm(float* a, int n, float* avg, float* disp);
__global__ void batch_norm_set(float* a, float* avg, float* disp, float g, float b);
__global__ void matrix_reverse(float* a, float* b, int n);
__global__ void matrix_convolution_multiplication(float* a, int m, float* b, int n, float* c);
__global__ void cnnPooling(float* a, int m, int n, float* b);
__global__ void pooling_backward(float* d, float* a, int m, float* b, int n, float* c);
__global__ void set_filter_changes(float* f, float* df, int n);
__global__ void make_padding_matrix(float* a, int n, float* b, int p);
__global__ void set_cnn_bias(float* a, int n, float* bias);
__global__ void set_cnn_bias_changes(float* b, float* db);

int main()
{
	//initialize
	//srand(static_cast<unsigned int>(time(NULL)));
	int i, j, k, size;

	images = new vector<float*>();
	labels = new vector<int>();
	read_image_label();

	bss = new float* [LEVEL_SIZE];
	wss = new float* [LEVEL_SIZE];

	//read_bias_weight();
	make_bias_weight();
	make_cnn_filter();

	mini_batch(TOTAL_SIZE, MINI_BATCH_SIZE);

	for (i = 0; i < FILTER_COUNT; i++) {
		hipMalloc((void**)&cnn[i], cnn_level_size[i + 1] * cnn_level_size[i + 1] * sizeof(float));
		hipMalloc((void**)&cnn_w[i], cnn_level_size[i + 1] * cnn_level_size[i + 1] * sizeof(float));
		hipMalloc((void**)&cnn_b[i], sizeof(float));
		hipMemset(cnn_b[i], 0, sizeof(float));
		hipMalloc((void**)&cnn_b_w[i], sizeof(float));

		hipMalloc((void**)&filter[i], FILTER_SIZE * sizeof(float));
		hipMemcpy(filter[i], filter_gen[i], FILTER_SIZE * sizeof(float), hipMemcpyHostToDevice);
		hipMalloc((void**)&df[i], FILTER_SIZE * sizeof(float));
		hipMalloc((void**)&fr[i], FILTER_SIZE * sizeof(float));

		pad_df_width = cnn_level_size[i + 1] + (FILTER_SIZE - 1) * 2;
		pad_df_size = pad_df_width * pad_df_width;
		hipMalloc((void**)&pad_df[i], pad_df_size * sizeof(float));
		hipMemset(pad_df[i], 0, pad_df_size * sizeof(float));
	}
	
	hipMalloc((void**)&avg, sizeof(float));
	hipMalloc((void**)&disp, sizeof(float));
	hipMalloc((void**)&level[0], level_size[0] * sizeof(float));
	hipMalloc((void**)&xssd[0], level_size[0] * sizeof(float));
	hipMalloc((void**)&output_dev, level_size[LEVEL_SIZE] * sizeof(float));
	output = new float[level_size[3]];
	ans = new float[level_size[3]];

	for (i = 0; i < LEVEL_SIZE; i++) {
		size = level_size[i + 1] * sizeof(float);
		hipMalloc((void**)&bss_dev[i], size);
		hipMemcpy(bss_dev[i], bss[i], size, hipMemcpyHostToDevice);
		hipMalloc((void**)&level[i + 1], size);
		hipMalloc((void**)&xssd[i + 1], size);

		size = level_size[i] * level_size[i + 1] * sizeof(float);
		hipMalloc((void**)&wss_dev[i], size);
		hipMemcpy(wss_dev[i], wss[i], size, hipMemcpyHostToDevice);
		hipMalloc((void**)&wsst[i], size);
		hipMalloc((void**)&wssd[i], size);
	}
	//~initialize

	//work
	for (i = 0; i < MINI_BATCH_SIZE; i++) {
		cout << "backprop " << i;

		//cout << endl;
		//for (j = 0; j < 28; j++) {
		//	for (k = 0; k < 28; k++) {
		//		cout << (images->at(mbatch->at(i))[j * 28 + k] > 0) << ' ';
		//	}
		//	cout << endl;
		//}

		predict(images->at(mbatch->at(i)), labels->at(mbatch->at(i)));
		cout << " cee : " << cee;
		backprop(images->at(mbatch->at(i)), labels->at(mbatch->at(i)));
		cout << " to " << cee << endl;
	}

	cnt = 0;
	correct = 0;
	for (i = 0; i < TOTAL_SIZE; i++) {
		predict(images->at(i), labels->at(i));
		cnt++;
		if (anstmp == labels->at(i)) {
			correct++;
		}
		cout << i << " predict : " << anstmp << " answer : " << labels->at(i) << " cee : " << cee << " data : ";
		for (j = 0; j < 10; j++) {
			//cout << output[j] << ' ';
			printf("%.4f ", output[j]);
		}
		cout << endl;
	}
	cout << "accuracy " << correct / (float)cnt << endl;
	//~work

	hipDeviceReset();

	return 0;
}

void read_image_label() {
	int header = 0, row = 0, col = 0, n, m, i = 0, j = 0, k = 0, l = 0, count = 0;

	//read image
	ifstream input_image(IMAGE_PATH, ios::binary);
	vector<char> bytes_i;
	char headerbuffer[IMAGE_HEADER_SIZE];
	input_image.read(headerbuffer, IMAGE_HEADER_SIZE);
	for (j = 0; j < IMAGE_HEADER_SIZE; j++) {
		bytes_i.push_back(headerbuffer[j]);
	}

	for (j = 0; j < TOTAL_SIZE; j++) {
		char* imagebuffer = new char[level_size[0]];
		input_image.read(imagebuffer, level_size[0]);
		for (k = 0; k < level_size[0]; k++) {
			bytes_i.push_back(imagebuffer[k]);
		}
	}

	//read label
	ifstream input_label(LABEL_PATH, ios::binary);
	vector<char> bytes_l;
	char labelbuffer[LABEL_HEADER_SIZE + TOTAL_SIZE];
	input_label.read(labelbuffer, LABEL_HEADER_SIZE + TOTAL_SIZE);
	for (j = 0; j < LABEL_HEADER_SIZE + TOTAL_SIZE; j++) {
		bytes_l.push_back(labelbuffer[j]);
	}

	n = bytes_i.size();
	m = bytes_l.size();

	cout << "image header : ";
	for (j = 0; j < IMAGE_HEADER_SIZE; j++) {
		cout << (int)(unsigned char)bytes_i[i++] << ' ';
	}
	cout << endl;

	cout << "label header : ";
	for (j = 0; j < LABEL_HEADER_SIZE; j++) {
		cout << (int)(unsigned char)bytes_l[l++] << ' ';
	}
	cout << endl;

	float* img;
	while (i < n && l < m && count < TOTAL_SIZE) {
		labels->push_back((int)(unsigned char)bytes_l[l++]);
		img = new float[level_size[0]];

		for (j = 0; j < level_size[0]; j++) {
			img[j] = (float)(unsigned char)bytes_i[i++] / 256.0;
		}
		images->push_back(img);
		count++;
	}

	cout << images->size() << " images" << endl;

	input_image.close();
	input_label.close();
}

void read_bias_weight() {
	ifstream bias_wieght(BIAS_WEIGHT_PATH);
	vector<char> chars(istreambuf_iterator<char>(bias_wieght), (istreambuf_iterator<char>()));
	int n = chars.size(), i, j, k, q = 0;
	vector<vector<float>*>* vs = new vector<vector<float>*>();
	vector<float>* v = new vector<float>();
	string s = "";
	for (i = 0; i < n; i++) {
		if (chars[i] == '\n') {
			vs->push_back(v);
			v = new vector<float>();
		}
		else if (chars[i] == '/') {
			v->push_back(stof(s));
			s.clear();
		}
		else {
			s += chars[i];
		}
	}

	for (i = 0; i < LEVEL_SIZE; i++) {
		bss[i] = &vs->at(i)->at(0);
		q++;
	}
	for (i = 0; i < LEVEL_SIZE; i++) {
		int size = level_size[i] * level_size[i + 1];
		float* x = new float[size];

		int pos = 0;
		for (j = 0; j < level_size[i]; j++) {
			memcpy(&x[pos], &vs->at(q++)->at(0), level_size[i + 1] * sizeof(float));
			pos += level_size[i + 1];
		}
		wss[i] = x;
	}
}

void make_bias_weight() {
	int i, j, size;
	float sd;
	float dist[10];

	for (i = 0; i < LEVEL_SIZE; i++) {
		bss[i] = new float[level_size[i + 1]];
		memset(bss[i], 0, level_size[i + 1] * sizeof(float));
		size = level_size[i] * level_size[i + 1];
		wss[i] = new float[size];
		sd = sqrt((ACT_FUNCTION + 1) / (float)level_size[i + 1]);
		normal_distribution<float> distribution(0.5, sd);
		default_random_engine generator;
		generator.seed(rand());
		memset(dist, 0, 10 * sizeof(float));
		for (j = 0; j < size; j++) {
			wss[i][j] = distribution(generator);
			if (wss[i][j] > 1) {
				wss[i][j] = 0.9999;
			}
			else if (wss[i][j] < 0) {
				wss[i][j] = 0.0001;
			}

			dist[(int)(wss[i][j] * 10)]++;
		}

		cout << "sd : " << sd << ", size : " << size << endl;
		for (j = 0; j < 10; j++) {
			cout << j * 0.1 << "~ : " << dist[j] << '(' << dist[j] * 100 / (float)size << "%)" << endl;
		}
		cout << endl;
	}
}

void make_cnn_filter() {
	int i, j;
	normal_distribution<float> distribution(0, 0.2);
	default_random_engine generator;
	generator.seed(rand());
	for (i = 0; i < FILTER_COUNT; i++) {
		filter_gen[i] = new float[FILTER_SIZE];
		for (j = 0; j < FILTER_SIZE; j++) {
			filter_gen[i][j] = distribution(generator);
			if (filter_gen[i][j] > 1) {
				filter_gen[i][j] = 1;
			}
			else if (filter_gen[i][j] < -1) {
				filter_gen[i][j] = -1;
			}
		}
	}
}

void mini_batch(int n, int count) {
	int i, x;
	mbatch = new vector<int>();
	int* arr = new int[n];
	for (i = 0; i < n; i++) {
		arr[i] = i;
	}
	for (i = n; i > n - count; i--) {
		x = rand() % i;
		mbatch->push_back(arr[x]);
		arr[x] = arr[i - 1];
	}
	delete[] arr;
}

void predict(float* image, int label) {
	int i;

	error = hipMemcpy(level[0], image, level_size[0] * sizeof(float), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		cout << "Faild copy image to device" << endl;
		return;
	}

	matrix_convolution_multiplication << <24, 24 >> > (level[0], 28, filter[0], 5, cnn[0]);	
	if (hipDeviceSynchronize() != hipSuccess) {
		cout << "error occured" << endl;
		return;
	}

	relu_cnn << <24, 24 >> > (cnn[0], 24, cnn_b[0]);
	if (hipDeviceSynchronize() != hipSuccess) {
		cout << "error occured" << endl;
		return;
	}

	matrix_convolution_multiplication << <20, 20 >> > (cnn[0], 24, filter[1], 5, cnn[1]);
	if (hipDeviceSynchronize() != hipSuccess) {
		cout << "error occured" << endl;
		return;
	}

	relu_cnn << <20, 20 >> > (cnn[1], 20, cnn_b[1]);
	if (hipDeviceSynchronize() != hipSuccess) {
		cout << "error occured" << endl;
		return;
	}

	cnnPooling << <10, 10 >> > (cnn[1], cnn_level_size[2], POOLING_SIZE, level[2]);
	if (hipDeviceSynchronize() != hipSuccess) {
		cout << "error occured" << endl;
		return;
	}

	//print_matrix << <1, 1 >> > (level[0], 28 * 28, 28);
	//print_matrix << <1, 1 >> > (filter[0], 5 * 5, 5);
	//print_matrix << <1, 1 >> > (cnn[0], 24 * 24, 24);
	//print_matrix << <1, 1 >> > (filter[1], 5 * 5, 5);
	//print_matrix << <1, 1 >> > (cnn[1], 20 * 20, 20);
	//print_matrix << <1, 1 >> > (level[2], 10 * 10, 10);

	//pass for cnn
	//for (i = 0; i < LEVEL_SIZE - 1; i++) {
	//	matrix_multiplication << <1, level_size[i + 1] >> > (level[i], 1, level_size[i], wss_dev[i], level_size[i], level_size[i + 1], level[i + 1]);
	//	//batch_norm<<<1,1>>>(level[i + 1], level_size[i + 1], avg, disp);		
	//	//batch_norm_set << <1, level_size[i + 1] >> > (level[i + 1], avg, disp, 1, 0);
	//	//print_matrix << <1, 1 >> > (level[i + 1], level_size[i + 1], 10);
	//	if (ACT_FUNCTION) {
	//		bias_relu << <1, level_size[i + 1] >> > (level[i + 1], bss_dev[i]);
	//	}
	//	else {
	//		bias_sigmoid << <1, level_size[i + 1] >> > (level[i + 1], bss_dev[i]);
	//	}
	//}

	matrix_multiplication << <1, level_size[LEVEL_SIZE] >> > (level[LEVEL_SIZE - 1], 1, level_size[LEVEL_SIZE - 1], wss_dev[LEVEL_SIZE - 1], level_size[LEVEL_SIZE - 1], level_size[LEVEL_SIZE], level[LEVEL_SIZE]);
	if (hipDeviceSynchronize() != hipSuccess) {
		cout << "error occured" << endl;
		return;
	}
	//print_matrix << <1, 1 >> > (level[3], 10, 10);
	softmax << <1, 1 >> > (level[LEVEL_SIZE], output_dev, level_size[LEVEL_SIZE]);
	if (hipDeviceSynchronize() != hipSuccess) {
		cout << "error occured" << endl;
		return;
	}
	//print_matrix << <1, 1 >> > (output_dev, 10, 10);
	hipMemcpy(output, output_dev, level_size[LEVEL_SIZE] * sizeof(float), hipMemcpyDeviceToHost);
	if (hipDeviceSynchronize() != hipSuccess) {
		cout << "error occured" << endl;
		return;
	}

	memset(ans, 0, level_size[LEVEL_SIZE] * sizeof(float));
	ans[label] = 1;

	cee = cross_entropy_error(output, ans, level_size[LEVEL_SIZE]);

	int maxidx = -1;
	float max = -1;
	for (i = 0; i < level_size[LEVEL_SIZE]; i++) {
		if (output[i] > max) {
			max = output[i];
			maxidx = i;
		}
	}

	anstmp = maxidx;
}

void backprop(float* img, int label) {
	int i, j, k;

	for (i = 0; i < LEARN_COUNT; i++) {
		predict(img, label);		
		
		//copy trans - matrix
		for (j = 0; j < LEVEL_SIZE; j++) {
			matrix_transpose << <level_size[j], level_size[j + 1] >> > (wss_dev[j], level_size[j], level_size[j + 1], wsst[j]);
			if (hipDeviceSynchronize() != hipSuccess) {
				cout << "error occured" << endl;
				return;
			}			
			print_matrix(level[j], level_size[j], 10, "level[j] " + to_string(i));
		}
		
		//copy reverse - matrix
		for (j = 0; j < FILTER_COUNT; j++) {
			matrix_reverse << <FILTER_WIDTH, FILTER_WIDTH >> > (filter[j], fr[j], FILTER_WIDTH);
			if (hipDeviceSynchronize() != hipSuccess) {
				cout << "error occured" << endl;
				return;
			}
		}

		//softmax - with - loss
		for (j = 0; j < level_size[LEVEL_SIZE]; j++) {
			output[j] -= ans[j];
		}

		//copy softmax - with - loss data to device
		error = hipMemcpy(xssd[LEVEL_SIZE], output, level_size[LEVEL_SIZE] * sizeof(float), hipMemcpyHostToDevice);
		if (error != hipSuccess) {
			cout << "Copy result to host faild" << endl;
			return;
		}
		print_matrix(xssd[LEVEL_SIZE], 10, 10, "xssd[LEVEL_SIZE] " + to_string(i));

		//set back to output
		for (j = 0; j < level_size[LEVEL_SIZE]; j++) {
			output[j] += ans[j];
		}
		

		//for (j = LEVEL_SIZE; j > 1; j--) { //ann
		for (j = LEVEL_SIZE; j > 2; j--) { //cnn
			matrix_multiplication << <1, level_size[j - 1] >> > (xssd[j], 1, level_size[j], wsst[j - 1], level_size[j], level_size[j - 1], xssd[j - 1]);
			if (hipDeviceSynchronize() != hipSuccess) {
				cout << "error occured" << endl;
				return;
			}
			print_matrix(xssd[2], 100, 10, "xssd[2] " + to_string(i));

			matrix_multiplication << <level_size[j - 1], level_size[j] >> > (level[j - 1], level_size[j - 1], 1, xssd[j], 1, level_size[j], wssd[j - 1]);
			if (hipDeviceSynchronize() != hipSuccess) {
				cout << "error occured" << endl;
				return;
			}
			print_matrix(wssd[2], 1000, 10, "wssd[2] " + to_string(i));

			if (ACT_FUNCTION) {
				relu_backward << <1, level_size[j - 1] >> > (xssd[j - 1], level[j - 1], 1, level_size[j - 1]);
				if (hipDeviceSynchronize() != hipSuccess) {
					cout << "error occured" << endl;
					return;
				}

			}
			else {
				sigmoid_backward << <1, level_size[j - 1] >> > (xssd[j - 1], level[j - 1], 1, level_size[j - 1]);
				if (hipDeviceSynchronize() != hipSuccess) {
					cout << "error occured" << endl;
					return;
				}
			}
			print_matrix(xssd[2], 100, 10, "xssd[2] " + to_string(i));
		}

		//ann
		//matrix_multiplication << <level_size[0], level_size[1] >> > (level[0], level_size[0], 1, xssd[1], 1, level_size[1], wssd[0]);
		//~ann

		//cnn
		pooling_backward << <20, 20 >> > (xssd[2], level[2], 10, cnn[1], 20, cnn_w[1]);
		if (hipDeviceSynchronize() != hipSuccess) {
			cout << "error occured" << endl;
			return;
		}
		print_matrix(cnn_w[1], 400, 20, "cnn_w[1] " + to_string(i));

		relu_backward << <20, 20 >> > (cnn_w[1], cnn[1], 20, 20);
		if (hipDeviceSynchronize() != hipSuccess) {
			cout << "error occured" << endl;
			return;
		}
		print_matrix(cnn_w[1], 400, 20, "cnn_w[1] " + to_string(i));

		//summary cnn_w[1]
		set_cnn_bias << <1, 1 >> > (cnn_w[1], 400, cnn_b_w[1]);

		matrix_convolution_multiplication << <FILTER_WIDTH, FILTER_WIDTH >> > (cnn[0], 24, cnn_w[1], 20, df[1]);
		if (hipDeviceSynchronize() != hipSuccess) {
			cout << "error occured" << endl;
			return;
		}
		print_matrix(df[1], 25, 5, "df[1] " + to_string(i));


		
		make_padding_matrix<<<20, 20>>>(cnn_w[1], 20, pad_df[1], FILTER_WIDTH - 1);
		if (hipDeviceSynchronize() != hipSuccess) {
			cout << "error occured" << endl;
			return;
		}
		print_matrix(pad_df[1], 28 * 28, 28, "pad_dfx[1] " + to_string(i));

		matrix_convolution_multiplication << <24, 24 >> > (pad_df[1], 28, fr[1], 5, cnn_w[0]);
		if (hipDeviceSynchronize() != hipSuccess) {
			cout << "error occured" << endl;
			return;
		}
		print_matrix(cnn_w[0], 24 * 24, 24, "cnn_w[0] " + to_string(i));
		
		relu_backward << <24, 24 >> > (cnn_w[0], cnn[0], 24, 24);
		if (hipDeviceSynchronize() != hipSuccess) {
			cout << "error occured" << endl;
			return;
		}
		print_matrix(cnn_w[0], 24 * 24, 24, "cnn_w[0] " + to_string(i));

		//average cnn_w[0]
		set_cnn_bias << <1, 1 >> > (cnn_w[0], 400, cnn_b_w[0]);

		matrix_convolution_multiplication << <FILTER_WIDTH, FILTER_WIDTH >> > (level[0], 28, cnn_w[0], 24, df[0]);
		if (hipDeviceSynchronize() != hipSuccess) {
			cout << "error occured" << endl;
			return;
		}
		print_matrix(df[0], 5 * 5, 5, "df[0] " + to_string(i));
		//~cnn

		//print_matrix << <1, 1 >> > (df[0], 5 * 5, 5);
		//print_matrix << <1, 1 >> > (df[1], 5 * 5, 5);

		//print_matrix << <1, 1 >> > (filter[0], 5 * 5, 5);
		//print_matrix << <1, 1 >> > (filter[1], 5 * 5, 5);

		//for (j = 0; j < LEVEL_SIZE; j++) { //ann
		for (j = 2; j < LEVEL_SIZE; j++) { // cnn
			set_weight_changes << < level_size[j], level_size[j + 1] >> > (wss_dev[j], wssd[j], level_size[j], level_size[j + 1]);
			if (hipDeviceSynchronize() != hipSuccess) {
				cout << "error occured" << endl;
				return;
			}

			set_bias_changes << <1, level_size[j + 1] >> > (bss_dev[j], xssd[j + 1], level_size[j + 1]);
			if (hipDeviceSynchronize() != hipSuccess) {
				cout << "error occured" << endl;
				return;
			}
		}

		//cnn
		for (j = 0; j < FILTER_COUNT; j++) {
			set_filter_changes << < FILTER_WIDTH, FILTER_WIDTH >> > (filter[j], df[j], FILTER_WIDTH);
			if (hipDeviceSynchronize() != hipSuccess) {
				cout << "error occured" << endl;
				return;
			}

			set_cnn_bias_changes << <1, 1 >> > (cnn_b[j], cnn_b_w[j]);
			if (hipDeviceSynchronize() != hipSuccess) {
				cout << "error occured" << endl;
				return;
			}
		}
	}
}

float cross_entropy_error(float* y, float* t, int n) {
	float res = 0;
	int i;

	for (i = 0; i < n; i++) {
		res += t[i] * log(y[i] + 1e-7);
	}

	return -res;
}

void print_matrix(float* a, int n, int r, string tag) {
	if (0
		|| tag.find("df[0]") != string::npos
		|| tag.find("df[1]") != string::npos
		) {
		float* buffer = new float[n];
		hipMemcpy(buffer, a, n * sizeof(float), hipMemcpyDeviceToHost);
		printf("\nprint %d items - %s\n", n, tag.c_str());
		int i;
		for (i = 0; i < n; i++) {
			if (i % r == 0 && i > 0) {
				printf("(%d)\n", i);
			}
			printf("%.4f ", buffer[i]);
		}
		printf("\n");
		delete buffer;
	}	
}

__global__ void softmax(float* a, float* b, int n) {
	int i;
	float max = 0, expsum = 0;
	for (i = 0; i < n; i++) {
		if (a[i] > max) {
			max = a[i];
		}
		b[i] = a[i];
	}
	for (i = 0; i < n; i++) {
		b[i] -= max;
		expsum += exp(b[i]);
	}

	for (i = 0; i < n; i++) {
		b[i] = exp(b[i]) / expsum;
	}
}
//row-col <<<m, p>>> matrix, {n == o} (m x n) x (o x p) = (m x p)
__global__ void matrix_multiplication(float* a, int m, int n, float* b, int o, int p, float* res) {
	int bi = blockIdx.x, ti = threadIdx.x, sb = bi * n, st = ti, c = bi * p + ti, i;
	res[c] = 0;
	for (i = 0; i < n; i++) {
		res[c] += a[sb + i] * b[st];
		st += p;
	}
}

__global__ void bias_sigmoid(float* a, float* bias) {
	int ti = threadIdx.x;
	a[ti] = 1 / (1 + exp(-(a[ti] + bias[ti])));
}

__global__ void bias_relu(float* a, float* bias) {
	int ti = threadIdx.x;
	a[ti] = a[ti] < 0 ? 0 : (a[ti] + bias[ti]);
}

__global__ void sigmoid_backward(float* dy, float* y, int m, int n) {
	int i = blockIdx.x, j = threadIdx.x, seq = n * i + j;
	float t = y[seq];
	dy[seq] *= t * (1 - t);
}

__global__ void relu_backward(float* dy, float* y, int m, int n) {
	int i = blockIdx.x, j = threadIdx.x, seq = n * i + j;
	dy[seq] *= y[seq] < 0 ? 0 : 1;
}

__global__ void relu_cnn(float* a, int n, float* bias) {	
	int bi = blockIdx.x, ti = threadIdx.x;
	int seq = bi * n + ti;
	a[seq] = a[seq] < 0 ? 0 : (a[seq] + *bias);
}

__global__ void set_weight_changes(float* ws, float* wds, int h, int w) {
	int bx = blockIdx.x;
	int tx = threadIdx.x;
	int seq = bx * w + tx;
	ws[seq] -= wds[seq] * LEARN_RATE;	
}

__global__ void set_bias_changes(float* bs, float* bds, int n) {
	int bx = blockIdx.x;
	int tx = threadIdx.x;
	bs[tx] -= bds[tx] * LEARN_RATE;
}

__global__ void matrix_transpose(float* a, int m, int n, float* b) {
	int j = blockIdx.x;
	int i = threadIdx.x;
	b[i * m + j] = a[j * n + i];
}

__global__ void batch_norm(float* a, int n, float* avg, float* disp) {
	int i;
	float tmp;

	for (i = 0; i < n; i++) {
		*avg += a[i];
	}
	*avg /= n;

	for (i = 0; i < n; i++) {
		tmp = a[i] - *avg;
		*disp += tmp * tmp;
	}
	*disp /= n;
}

__global__ void batch_norm_set(float* a, float* avg, float* disp, float g, float b) {
	int i = threadIdx.x;
	a[i] = g * ((a[i] - *avg) / sqrt(*disp * *disp + 10e-7)) + b;
}

__global__ void matrix_reverse(float* a, float* b, int n) {
	int bl = blockIdx.x, th = threadIdx.x;
	int idx = bl * n + th;
	int ridx = (n - bl - 1) * n + (n - th - 1);
	b[idx] = a[ridx];
}

__global__ void matrix_convolution_multiplication(float* a, int m, float* b, int n, float* c) {
	int bl = blockIdx.x, th = threadIdx.x;
	int o = m - n + 1, p = bl * o + th;
	int i, j;
	c[p] = 0;
	for (i = bl; i < bl + n; i++) {
		for (j = th; j < th + n; j++) {
			c[p] += a[i * m + j] * b[(i - bl) * n + (j - th)];
		}
	}
}

__global__ void cnnPooling(float* a, int m, int n, float* b) {
	int bl = blockIdx.x, th = threadIdx.x;
	int bs = bl * n, ts = th * n, rs = bl * (m / n) + th, tmp;
	int i, j;	
	b[rs] = 0;
	for (i = bs; i < bs + n; i++) {
		for (j = ts; j < ts + n; j++) {
			tmp = i * m + j;
			b[rs] = a[tmp] > b[rs] ? a[tmp] : b[rs];
		}
	}
}

//d : delta, a : pooling data, m : pooling size, b : cnn data, n : cnn size, c : pooling backward result
__global__ void pooling_backward(float* d, float* a, int m, float* b, int n, float* c) {
	int bl = blockIdx.x, th = threadIdx.x;
	int cnnIdx = bl * n + th;
	int poolIdx = (bl / 2) * m + (th / 2);
	c[cnnIdx] = a[poolIdx] == b[cnnIdx] ? d[poolIdx] : 0;
}

__global__ void set_filter_changes(float* f, float* df, int n) {
	int bl = blockIdx.x, th = threadIdx.x;
	int idx = bl * n + th;	
	f[idx] -= df[idx] * LEARN_RATE;
}

//a : base, n :base width, b : result, p : padding size
__global__ void make_padding_matrix(float* a, int n, float* b, int p) {
	int bl = blockIdx.x, th = threadIdx.x;
	int idx = bl * n + th;
	int pad_width = n + 2 * p;
	int pad_bl = p - 1 + bl, pad_th = p - 1 + th;
	int pad_idx = pad_bl * pad_width + pad_th;
	b[pad_idx] = a[idx];
}

__global__ void set_cnn_bias(float* a, int n, float* bias) {
	int i;
	*bias = 0;
	for (i = 0; i < n; i++) {
		*bias += a[i];
	}	
}

__global__ void set_cnn_bias_changes(float* b, float* db) {
	*b -= *db * LEARN_RATE;
	//printf("bias : %f\n", *db);
}