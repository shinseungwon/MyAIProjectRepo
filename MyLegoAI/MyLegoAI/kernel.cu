#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <vector>
#include <random>
#include <fstream>
#include <time.h>

using namespace std;

//prepare test data
#define IMAGE_PATH "mnist\\train-images.idx3-ubyte"
#define LABEL_PATH "mnist\\train-labels.idx1-ubyte"

#define BIAS_WEIGHT_PATH "mnist\\biasweight.txt" //learned data

#define IMAGE_HEADER_SIZE 16
#define LABEL_HEADER_SIZE 8

#define TOTAL_SIZE 100 // <= 60000
#define MINI_BATCH_SIZE 100 // <= TOTAL_IMG_SIZE

#define INPUT_SIZE 784
//~prepare test data

#define LEARN_COUNT 10

enum layer_type { AFF, CONV, POOLING };

void read_image_label();
void mini_batch(int n, int count);

__global__ void update_delta(int n, float* weight, float* delta, float lr);
__global__ void add_bias(int n, float* a, float* bias);
__global__ void add_bias_array(int n, float* a, float* bias);
__global__ void relu(int n, float* a);
__global__ void relu_backward(float* dy, float* y, int m, int n);
__global__ void pooling(float* a, int m, int n, float* b);
__global__ void pooling_backward(float* d, float* a, int m, float* b, int n, float* c);
__global__ void matrix_multiplication(float* a, int m, int n, float* b, int o, int p, float* res);
__global__ void matrix_convolution_multiplication(float* a, int m, float* b, int n, float* c);
__global__ void matrix_transpose(float* a, int m, int n, float* b);
__global__ void matrix_reverse(float* a, float* b, int n);
__global__ void make_padding_matrix(float* a, int n, float* b, int p);
__global__ void set_weight_changes(float* ws, float* wds, int h, int w, float lr);
__global__ void set_bias_changes(float* bs, float* bds, int n, float lr);
__global__ void batch_norm(float* a, float avg, float disp, float g, float b);

class layer {
public:
	layer_type type;
	float* data;
	float* weight;
	float* bias = 0;

	float* data_grad;
	float* weight_grad;
	float* bias_grad;

	int data_size = 0;
	int weight_size = 0;
	int x;
	int y;
	int z;
	int w;

	float lr = 0.1;

	//batch_normalization
	float dst_avg = 0;
	float dst_dist = 1;
	float g = 1;
	float dg = 0;
	float b = 0;
	float db = 0;

	//output
	float* output_1;
	float* output_2;
	float* output_dev;
	float* answer;
	float* output_grad;

	layer(layer_type type, int x, int y, int z, int w) {
		this->type = type;
		this->x = x;
		this->y = y;
		this->z = z;
		this->w = w;
		this->data_size = x * y;
		this->weight_size = z * w;

		hipError_t error;

		error = hipMalloc((void**)&data, data_size * sizeof(float));
		if (error != hipSuccess) {
			throw exception(hipGetErrorString(error));
		}

		error = hipMemset(data, 0, data_size * sizeof(float));
		if (error != hipSuccess) {
			throw exception(hipGetErrorString(error));
		}

		error = hipMalloc((void**)&data_grad, data_size * sizeof(float));
		if (error != hipSuccess) {
			throw exception(hipGetErrorString(error));
		}

		error = hipMemset(data_grad, 0, data_size * sizeof(float));
		if (error != hipSuccess) {
			throw exception(hipGetErrorString(error));
		}

		error = hipMalloc((void**)&weight, weight_size * sizeof(float));
		if (error != hipSuccess) {
			throw exception(hipGetErrorString(error));
		}

		error = hipMemset(weight, 0, weight_size * sizeof(float));
		if (error != hipSuccess) {
			throw exception(hipGetErrorString(error));
		}

		error = hipMalloc((void**)&weight_grad, weight_size * sizeof(float));
		if (error != hipSuccess) {
			throw exception(hipGetErrorString(error));
		}

		error = hipMemset(weight_grad, 0, weight_size * sizeof(float));
		if (error != hipSuccess) {
			throw exception(hipGetErrorString(error));
		}

		int bias_size;

		if (type == AFF) {
			bias_size = data_size * sizeof(float);
		}
		else if (type == CONV) {
			bias_size = sizeof(float);
		}
		else if (type == POOLING) {
			bias_size = 0;
		}

		error = hipMalloc((void**)&bias, bias_size);
		if (error != hipSuccess) {
			throw exception(hipGetErrorString(error));
		}

		error = hipMemset(bias, 0, bias_size);
		if (error != hipSuccess) {
			throw exception(hipGetErrorString(error));
		}

		error = hipMalloc((void**)&bias_grad, bias_size);
		if (error != hipSuccess) {
			throw exception(hipGetErrorString(error));
		}

		error = hipMemset(bias_grad, 0, bias_size);
		if (error != hipSuccess) {
			throw exception(hipGetErrorString(error));
		}
	}

	void set_data(float* data, int size) {
		if (this->data_size != size) {
			throw exception("Size not match");
		}
		hipError_t error = hipMemcpy(this->data, data, size * sizeof(float), hipMemcpyDeviceToDevice);
		if (error != hipSuccess) {
			cout << "set_data error" << endl;
			throw exception(hipGetErrorString(error));
		}
	}

	void generate_weight(float mean, float dist) {
		normal_distribution<float> distribution(mean, dist);
		default_random_engine generator;
		generator.seed(rand());
		float* weight_gen = new float[weight_size];
		for (int i = 0; i < weight_size; i++) {
			weight_gen[i] = distribution(generator);
		}
		hipError_t error = hipMemcpy(weight, weight_gen, weight_size * sizeof(float), hipMemcpyHostToDevice);
		if (error != hipSuccess) {
			cout << "generate_weight error" << endl;
			throw exception(hipGetErrorString(error));
		}
	}

	void change_weight(float* changes, float lr) {
		update_delta << <this->z, this->w >> > (weight_size, weight, changes, lr);
		hipError_t error = hipDeviceSynchronize();
		if (error != hipSuccess) {
			cout << "error occured" << endl;
			throw exception(hipGetErrorString(error));
		}
	}

	void change_bias(float* changes, float lr) {
		int bias_size;
		if (this->type == AFF) {
			bias_size = data_size;
		}
		else if (this->type == CONV) {
			bias_size = 1;
		}
		update_delta << <1, bias_size >> > (1, bias, changes, lr);
		hipError_t error = hipDeviceSynchronize();
		if (error != hipSuccess) {
			cout << "error occured" << endl;
			throw exception(hipGetErrorString(error));
		}
	}

	void forward(float* dst) {
		hipError_t error;
		if (this->type == AFF) {
			matrix_multiplication << <1, w >> > (data, this->x, this->y, weight, this->z, this->w, dst);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			add_bias_array << <1, y >> > (y, dst, bias);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			//batchn			
			set_avg_dist(dst, w);
			batch_norm << <1, w >> > (dst, dst_avg, dst_dist, g, b);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}
			//~batchn

			relu << <1, y >> > (y, dst);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}
		}
		else if (this->type == CONV) {
			int dst_width = x - z + 1;
			matrix_convolution_multiplication << <dst_width, dst_width >> > (data, x, weight, z, dst);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			add_bias << <dst_width, dst_width >> > (dst_width, dst, bias);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			//batchn
			int dst_size = dst_width * dst_width;
			set_avg_dist(dst, dst_size);
			batch_norm << <1, dst_size >> > (dst, dst_avg, dst_dist, g, b);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}
			//~batchn

			relu << <dst_width, dst_width >> > (dst_width, dst);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}
		}
		else if (this->type == POOLING) {
			int pooling_size = x / z;
			pooling << <pooling_size, pooling_size >> > (data, x, z, dst);
			if (hipDeviceSynchronize() != hipSuccess) {
				cout << "error occured" << endl;
				return;
			}
		}
	}

	void backward(float* delta, float* _data = nullptr) {
		hipError_t error;
		if (this->type == AFF) {
			relu_backward << <this->x, this->y >> > (data_grad, data, x, y);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			batch_norm_backward(data_grad, _data, w);

			float* weight_trans;
			error = hipMalloc((void**)&weight_trans, weight_size * sizeof(float));
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			matrix_transpose << <z, w >> > (weight, z, w, weight_trans);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			matrix_multiplication << <1, data_size >> > (delta, 1, w, weight_trans, w, z, data_grad);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			matrix_multiplication << <z, w >> > (data, y, 1, delta, 1, w, weight_grad);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			set_weight_changes << <z, w >> > (weight, weight_grad, z, w, this->lr);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			set_bias_changes << <1, y >> > (bias, delta, w, this->lr);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			error = hipFree(weight_trans);
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}
		}
		else if (this->type == CONV) {
			relu_backward << <this->x, this->y >> > (data_grad, data, x, y);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}
			
			int delta_width = x - z + 1;
			int delta_size = delta_width * delta_width;
			float* delta_host = new float[delta_size];
			batch_norm_backward(data_grad, _data, delta_size);

			//1. get weight grad
			error = hipMemcpy(delta_host, delta, delta_size * sizeof(float), hipMemcpyDeviceToHost);
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			float bias_host = 0;
			for (int i = 0; i < delta_size; i++) {
				bias_host += delta_host[i];
			}
			error = hipMemcpy(bias_grad, &bias_host, sizeof(float), hipMemcpyHostToDevice);
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			matrix_convolution_multiplication << <z, w >> > (data, x, delta, delta_width, weight_grad);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			//2. apply changes
			set_weight_changes << <z, w >> > (weight, weight_grad, z, w, this->lr);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			//3. perpare padding matrix
			int padding_matrix_width = delta_width + 2 * (z - 1);
			int padding_matrix_size = padding_matrix_width * padding_matrix_width;
			float* padding_matrix;
			error = hipMalloc((void**)&padding_matrix, padding_matrix_size * sizeof(float));
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			error = hipMemset(padding_matrix, 0, padding_matrix_size * sizeof(float));
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			make_padding_matrix << <delta_width, delta_width >> > (delta, delta_width, padding_matrix, z - 1);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			//4. prepare reverse matrix
			float* weight_reverse_matrix;
			error = hipMalloc((void**)&weight_reverse_matrix, weight_size * sizeof(float));
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			matrix_reverse << <z, w >> > (weight, weight_reverse_matrix, z);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			//5. cnn 3 and 4
			matrix_convolution_multiplication << <x, x >> > (padding_matrix, padding_matrix_width, weight_reverse_matrix, z, data_grad);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}
		}
		else if (this->type == POOLING) {
			int pooling_size = x / z;
			pooling_backward << <x, y >> > (delta, _data, pooling_size, data, x, weight);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}
		}
	}

	void set_result() {

		if (this->type == AFF) {
			hipError_t error;			
			hipMalloc((void**)&output_dev, data_size * sizeof(float));
			matrix_multiplication << <1, w >> > (data, this->x, this->y, weight, this->z, this->w, output_dev);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			add_bias_array << <1, y >> > (y, output_dev, bias);
			error = hipDeviceSynchronize();
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			int output_size = w;
			output_1 = new float[output_size];
			output_2 = new float[output_size];
			error = hipMemcpy(output_1, output_dev, output_size * sizeof(float), hipMemcpyDeviceToHost);
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}

			softmax(output_1, output_2, output_size);

			error = hipMemcpy(output_dev, output_2, output_size * sizeof(float), hipMemcpyHostToDevice);
			if (error != hipSuccess) {
				cout << "error occured" << endl;
				throw exception(hipGetErrorString(error));
			}
		}
		else {
			throw exception("Only affine layer gets result");
		}
	}

	void set_answer(float* answer) {
		this->answer = answer;
	}

	void softmax(float* a, float* b, int n) {
		int i;
		float max = 0, expsum = 0;
		for (i = 0; i < n; i++) {
			if (a[i] > max) {
				max = a[i];
			}
			b[i] = a[i];
		}
		for (i = 0; i < n; i++) {
			b[i] -= max;
			expsum += exp(b[i]);
		}

		for (i = 0; i < n; i++) {
			b[i] = exp(b[i]) / expsum;
		}
	}

	float cross_entrophy_error() {
		float res = 0;
		int i, output_size = w;

		for (i = 0; i < output_size; i++) {
			res += answer[i] * log(output_2[i] + 1e-7);
		}

		return -res;
	}

	void result_backward() {
		int result_size = w;
		hipError_t error = hipMalloc((void**)&output_grad, result_size * sizeof(float));
		if (error != hipSuccess) {
			cout << "error occured" << endl;
			throw exception(hipGetErrorString(error));
		}

		float* dy = new float[result_size];
		for (int i = 0; i < result_size; i++) {
			dy[i] = output_2[i] - answer[i];
		}
		hipMemcpy(output_grad, dy, result_size * sizeof(float), hipMemcpyHostToDevice);
	}

	void set_avg_dist(float* dst, int size) {
		float* data_host = new float[size];
		hipError_t error = hipMemcpy(data_host, dst, size * sizeof(float), hipMemcpyDeviceToHost);

		dst_avg = 0;
		for (int i = 0; i < size; i++) {
			dst_avg += data_host[i];
		}
		dst_avg /= size;

		dst_dist = 1;
		for (int i = 0; i < size; i++) {
			float tmp = data_host[i] - dst_avg;
			dst_dist += tmp * tmp;
		}
		dst_dist /= size;

		delete data_host;
	}

	void batch_norm_backward(float* delta, float* _data, int size) {
		hipError_t error;
		int i, j, k;

		float* delta_host = new float[size];
		error = hipMemcpy(delta_host, delta, size * sizeof(float), hipMemcpyDeviceToHost);
		if (error != hipSuccess) {
			cout << "error occured" << endl;
			throw exception(hipGetErrorString(error));
		}

		float* data_host = new float[size];
		error = hipMemcpy(data_host, _data, size * sizeof(float), hipMemcpyDeviceToHost);
		if (error != hipSuccess) {
			cout << "error occured" << endl;
			throw exception(hipGetErrorString(error));
		}

		db = 0;
		for (i = 0; i < size; i++) {
			db += delta_host[i];
		}

		dg = 0;
		for (i = 0; i < size; i++) {
			dg += (data_host[i] - b) / g * delta_host[i];
		}

		float* dxu1 = new float[size];
		float* divar = new float[size]; //ivar = dist^2 - 10e-7
		float ivar = 1 / sqrt(dst_dist * dst_dist - 10e-7);
		for (i = 0; i < size; i++) {
			dxu1[i] = delta_host[i] * ivar;
			divar[i] = delta_host[i] * (data_host[i] - dst_avg);
		}

		float* dsqrtvar = new float[size];
		float sqrtvar = 1 / ivar;
		for (i = 0; i < size; i++) {
			dsqrtvar[i] = divar[i] * (-1 / sqrtvar * sqrtvar);
		}

		float* dvar = new float[size];
		float var = dst_dist * dst_dist;
		for (i = 0; i < size; i++) {
			dvar[i] = 0.5 * dsqrtvar[i] / sqrt(var + 10e-7);
		}

		float* dsq = new float[size];
		for (i = 0; i < size; i++) {
			dsq[i] = dvar[i];
		}

		float* dxu2 = new float[size];
		float xu = dst_dist;
		for (i = 0; i < size; i++) {
			dxu2[i] = 2 * xu * dsq[i];
		}

		float* dx1 = new float[size];
		float* du = new float[size];
		for (i = 0; i < size; i++) {
			dx1[i] = dxu1[i] + dxu2[i];
			du[i] = -dx1[i];
		}

		float* dx2 = new float[size];
		for (i = 0; i < size; i++) {
			dx2[i] = du[i];
		}

		float* dx = new float[size];
		for (i = 0; i < size; i++) {
			dx[i] = dx1[i] + dx2[i];
		}

		for (i = 0; i < size; i++) {
			delta_host[i] = dx[i];
		}

		error = hipMemcpy(delta, delta_host, size * sizeof(float), hipMemcpyHostToDevice);
		if (error != hipSuccess) {
			cout << "error occured" << endl;
			throw exception(hipGetErrorString(error));
		}

		b += db * lr;
		g += dg * lr;

		delete delta_host;
		delete data_host;
		delete dxu1;
		delete divar;
		delete dsqrtvar;
		delete dvar;
		delete dsq;
		delete dxu2;
		delete dx1;
		delete du;
		delete dx2;
		delete dx;
	}

	void print_data(int lb = 10, bool grad = 0) {
		float* tmp = new float[data_size];
		hipError_t error;
		if (grad) {
			error = hipMemcpy(tmp, data_grad, data_size * sizeof(float), hipMemcpyDeviceToHost);
		}
		else {
			error = hipMemcpy(tmp, data, data_size * sizeof(float), hipMemcpyDeviceToHost);
		}

		if (error != hipSuccess) {
			cout << "print_data copy failed" << endl;
			throw exception(hipGetErrorString(error));
		}

		printf("%d items\n", data_size);
		printf("----------------------------------------------------------------------------------------------------\n");
		for (int i = 0; i < data_size; i++) {
			if (i % lb == 0 && i != 0) {
				printf("(%d)\n", i);
			}
			printf("%.4f ", tmp[i]);
		}
		printf("(%d)\n----------------------------------------------------------------------------------------------------\n", data_size);
		delete tmp;
	}

	void print_weight(int lb = 10, bool grad = 0) {
		float* tmp = new float[weight_size];
		hipError_t error;
		if (grad) {
			error = hipMemcpy(tmp, weight_grad, weight_size * sizeof(float), hipMemcpyDeviceToHost);
		}
		else {
			error = hipMemcpy(tmp, weight, weight_size * sizeof(float), hipMemcpyDeviceToHost);
		}

		if (error != hipSuccess) {
			cout << "print_weight copy failed" << endl;
			throw exception(hipGetErrorString(error));
		}

		printf("%d items\n", weight_size);
		printf("----------------------------------------------------------------------------------------------------\n");
		for (int i = 0; i < weight_size; i++) {
			if (i % lb == 0 && i != 0) {
				printf("(%d)\n", i);
			}
			printf("%.4f ", tmp[i]);
		}
		printf("(%d)\n----------------------------------------------------------------------------------------------------\n", weight_size);
		delete tmp;
	}
};

//variables
vector<float*>* images;
vector<int>* labels;
vector<int>* mbatch;

int main()
{
	srand(static_cast<unsigned int>(time(NULL)));
	read_image_label();

	//input(784 - 28 * 28) - conv(5 - 24 * 24) - conv(5 - 20 * 20) - aff(400 * 100 - 1 * 100) - aff(100 * 10 - 1 * 10) - result

	int i, j, k;
	layer* input = new layer(CONV, 28, 28, 5, 5);
	input->generate_weight(0.5, sqrt(2 / 50.0));
	layer* conv1 = new layer(CONV, 24, 24, 5, 5);
	conv1->generate_weight(0.5, sqrt(2 / 50.0));

	//layer* aff1 = new layer(AFF, 1, 400, 400, 100);
	//aff1->generate_weight(0.5, sqrt(1 / 50.0));
	layer* pool1 = new layer(POOLING, 20, 20, 2, 2);

	layer* output = new layer(AFF, 1, 100, 100, 10);
	output->generate_weight(0.5, sqrt(1 / 50.0));

	float* answer;
	float cross_entrophy_error;
	for (i = 0; i < images->size(); i++) {
		input->set_data(images->at(i), 784); //set input

		//set answer
		answer = new float[10];
		memset(answer, 0, 10 * sizeof(float));
		answer[labels->at(i)] = 1;
		output->set_answer(answer);
		//~set answer

		for (j = 0; j < LEARN_COUNT; j++) {
			//predict
			input->forward(conv1->data);
			//input->print_data(28);

			//conv1->forward(aff1->data);
			//aff1->forward(output->data);
			conv1->forward(pool1->data);
			//conv1->print_data(24);

			pool1->forward(output->data);
			//pool1->print_data(20);

			output->set_result();
			//output->print_data();

			//for (k = 0; k < 10; k++) {
			//	cout << output->output_1[k] << ' ';
			//}
			//cout << endl;

			//for (k = 0; k < 10; k++) {
			//	cout << output->output_2[k] << ' ';
			//}
			//cout << endl;

			if (j == 0) {
				cross_entrophy_error = output->cross_entrophy_error();
				cout << i << " - cross entrophy error : " << cross_entrophy_error;
			}
			else if (j == LEARN_COUNT - 1) {
				cross_entrophy_error = output->cross_entrophy_error();
				cout << " to " << cross_entrophy_error << endl;
			}
			//~predict		

			output->result_backward();
			output->backward(output->output_grad, output->output_dev);
			//output->print_weight(10, 1);
			//output->print_weight(10, 0);

			//aff1->backward(output->data_grad);
			//aff1->print_weight(10, 1);

			pool1->backward(output->data_grad, output->data);

			//conv1->backward(aff1->data_grad);
			conv1->backward(pool1->data_grad, pool1->data);
			//conv1->print_weight(10, 0);
			//conv1->print_weight(10, 1);

			input->backward(conv1->data_grad, conv1->data);
			//input->print_weight(10, 0);
			//input->print_weight(10, 1);
		}
	}

	mini_batch(TOTAL_SIZE, MINI_BATCH_SIZE);
	int correct = 0;

	for (i = 0; i < mbatch->size(); i++) {
		input->set_data(images->at(mbatch->at(i)), 784); //set input
		input->forward(conv1->data);
		conv1->forward(pool1->data);
		pool1->forward(output->data);
		output->set_result();
		float max = 0;
		int max_idx = -1;
		for (j = 0; j < 10; j++) {
			if (output->output_2[j] > max) {
				max = output->output_2[j];
				max_idx = j;
			}
		}
		cout << "predict : " << max_idx << " answer : " << labels->at(mbatch->at(i)) << endl;
		if (labels->at(mbatch->at(i)) == max_idx) {
			correct++;
		}
	}
	float total = TOTAL_SIZE;
	cout << "accuracy : " << correct / total * 100 << '%' << endl;

	hipDeviceReset();

	return 0;
}

void read_image_label() {
	images = new vector<float*>();
	labels = new vector<int>();

	int header = 0, row = 0, col = 0, n, m, i = 0, j = 0, k = 0, l = 0, count = 0;

	//read image
	ifstream input_image(IMAGE_PATH, ios::binary);
	vector<char> bytes_i;
	char headerbuffer[IMAGE_HEADER_SIZE];
	input_image.read(headerbuffer, IMAGE_HEADER_SIZE);
	for (j = 0; j < IMAGE_HEADER_SIZE; j++) {
		bytes_i.push_back(headerbuffer[j]);
	}

	for (j = 0; j < TOTAL_SIZE; j++) {
		char* imagebuffer = new char[INPUT_SIZE];
		input_image.read(imagebuffer, INPUT_SIZE);
		for (k = 0; k < INPUT_SIZE; k++) {
			bytes_i.push_back(imagebuffer[k]);
		}
	}

	//read label
	ifstream input_label(LABEL_PATH, ios::binary);
	vector<char> bytes_l;
	char labelbuffer[LABEL_HEADER_SIZE + TOTAL_SIZE];
	input_label.read(labelbuffer, LABEL_HEADER_SIZE + TOTAL_SIZE);
	for (j = 0; j < LABEL_HEADER_SIZE + TOTAL_SIZE; j++) {
		bytes_l.push_back(labelbuffer[j]);
	}

	n = bytes_i.size();
	m = bytes_l.size();

	cout << "image header : ";
	for (j = 0; j < IMAGE_HEADER_SIZE; j++) {
		cout << (int)(unsigned char)bytes_i[i++] << ' ';
	}
	cout << endl;

	cout << "label header : ";
	for (j = 0; j < LABEL_HEADER_SIZE; j++) {
		cout << (int)(unsigned char)bytes_l[l++] << ' ';
	}
	cout << endl;

	float* img;
	while (i < n && l < m && count < TOTAL_SIZE) {
		labels->push_back((int)(unsigned char)bytes_l[l++]);
		img = new float[INPUT_SIZE];

		for (j = 0; j < INPUT_SIZE; j++) {
			img[j] = (float)(unsigned char)bytes_i[i++] / 256.0;
		}
		float* imgDev;
		hipMalloc((void**)&imgDev, INPUT_SIZE * sizeof(float));
		hipMemcpy(imgDev, img, INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice);
		images->push_back(imgDev);
		count++;
	}

	cout << images->size() << " images" << endl;

	input_image.close();
	input_label.close();
}

void mini_batch(int n, int count) {
	int i, x;
	mbatch = new vector<int>();
	int* arr = new int[n];
	for (i = 0; i < n; i++) {
		arr[i] = i;
	}
	for (i = n; i > n - count; i--) {
		x = rand() % i;
		mbatch->push_back(arr[x]);
		arr[x] = arr[i - 1];
	}
	delete[] arr;
}



//n : col count ( thread count )
__global__ void update_delta(int n, float* weight, float* delta, float lr) {
	int idx = blockIdx.x * n + threadIdx.x;
	weight[idx] -= lr * delta[idx];
}

__global__ void add_bias(int n, float* a, float* bias) {
	int idx = blockIdx.x * n + threadIdx.x;	
	a[idx] += *bias;
}

__global__ void add_bias_array(int n, float* a, float* bias) {
	int idx = blockIdx.x * n + threadIdx.x;
	a[idx] += bias[idx];
}

__global__ void relu(int n, float* a) {
	int idx = blockIdx.x * n + threadIdx.x;
	if (a[idx] < 0) a[idx] = 0;
}

__global__ void relu_backward(float* dy, float* y, int m, int n) {
	int i = blockIdx.x, j = threadIdx.x, seq = n * i + j;
	dy[seq] *= y[seq] < 0 ? 0 : 1;
}

__global__ void pooling(float* a, int m, int n, float* b) {
	int bl = blockIdx.x, th = threadIdx.x;
	int bs = bl * n, ts = th * n, rs = bl * (m / n) + th, tmp;
	int i, j;
	b[rs] = 0;
	for (i = bs; i < bs + n; i++) {
		for (j = ts; j < ts + n; j++) {
			tmp = i * m + j;
			b[rs] = a[tmp] > b[rs] ? a[tmp] : b[rs];
		}
	}
}

//d : delta, a : pooling data, m : pooling size, b : cnn data, n : cnn size, c : pooling backward result
__global__ void pooling_backward(float* d, float* a, int m, float* b, int n, float* c) {
	int bl = blockIdx.x, th = threadIdx.x;
	int cnnIdx = bl * n + th;
	int poolIdx = (bl / 2) * m + (th / 2);
	c[cnnIdx] = a[poolIdx] == b[cnnIdx] ? d[poolIdx] : 0;
}

//<<<m, p>>> matrix, {n == o} (m x n) x (o x p) = (m x p)
__global__ void matrix_multiplication(float* a, int m, int n, float* b, int o, int p, float* res) {
	int bi = blockIdx.x, ti = threadIdx.x, sb = bi * n, st = ti, c = bi * p + ti, i;
	res[c] = 0;
	for (i = 0; i < n; i++) {
		res[c] += a[sb + i] * b[st];
		st += p;
	}
}

//<<<m - n + 1, m - n + 1>>>
__global__ void matrix_convolution_multiplication(float* a, int m, float* b, int n, float* c) {
	int bl = blockIdx.x, th = threadIdx.x;
	int o = m - n + 1, p = bl * o + th;
	int i, j;
	c[p] = 0;
	for (i = bl; i < bl + n; i++) {
		for (j = th; j < th + n; j++) {
			c[p] += a[i * m + j] * b[(i - bl) * n + (j - th)];
		}
	}
}

__global__ void matrix_transpose(float* a, int m, int n, float* b) {
	int j = blockIdx.x;
	int i = threadIdx.x;
	b[i * m + j] = a[j * n + i];
}

__global__ void matrix_reverse(float* a, float* b, int n) {
	int bl = blockIdx.x, th = threadIdx.x;
	int idx = bl * n + th;
	int ridx = (n - bl - 1) * n + (n - th - 1);
	b[idx] = a[ridx];
}

__global__ void make_padding_matrix(float* a, int n, float* b, int p) {
	int bl = blockIdx.x, th = threadIdx.x;
	int idx = bl * n + th;
	int pad_width = n + 2 * p;
	int pad_bl = p - 1 + bl, pad_th = p - 1 + th;
	int pad_idx = pad_bl * pad_width + pad_th;
	b[pad_idx] = a[idx];
}

__global__ void set_weight_changes(float* ws, float* wds, int h, int w, float lr) {
	int bx = blockIdx.x;
	int tx = threadIdx.x;
	int seq = bx * w + tx;
	ws[seq] -= wds[seq] * lr;
}

__global__ void set_bias_changes(float* bs, float* bds, int n, float lr) {
	int bx = blockIdx.x;
	int tx = threadIdx.x;
	bs[tx] -= bds[tx] * lr;
}

__global__ void batch_norm(float* a, float avg, float disp, float g, float b) {
	int i = threadIdx.x;
	a[i] = g * ((a[i] - avg) / sqrt(disp * disp + 10e-7)) + b;
}